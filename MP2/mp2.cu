#include "hip/hip_runtime.h"


// Compute C = A * B TODO
__global__ void matrixMultiply(float *A, float *B, float *C, int numARows,
                               int numAColumns, int numBRows,
                               int numBColumns, int numCRows,
                               int numCColumns)
{
    //@@ Insert code to implement matrix multiplication here
}
#define BLOCK_WIDTH 256
int main(int argc, char **argv)
{
    float *hostA; // The A matrix
    float *hostB; // The B matrix
    float *hostC; // The output C matrix
    float *deviceA;
    float *deviceB;
    float *deviceC;
    int numARows;    // number of rows in the matrix A
    int numAColumns; // number of columns in the matrix A
    int numBRows;    // number of rows in the matrix B
    int numBColumns; // number of columns in the matrix B
    int numCRows;    // number of rows in the matrix C (you have to set this)
    int numCColumns; // number of columns in the matrix C (you have to set
                     // this)

    // TODO replace with custom data loading to hostA and hostB
    //@@ Set numCRows and numCColumns DONE
    numCRows = numARows;
    numCColumns = numBColumns;
    //@@ Allocate the hostC matrix DONE
    hostc = malloc(numARows * numBColumns * sizeof(float));

    //@@ Allocate GPU memory here DONE
    hipMalloc((void **)&deviceA, numARows * numAColumns * sizeof(float));
    hipMalloc((void **)&deviceB, numBRows * numBColumns * sizeof(float));
    hipMalloc((void **)&deviceC, numCRows * numCColumns * sizeof(float));

    //@@ Copy memory to the GPU here DONE
    hipMemcpy(deviceA, hostA, numARows * numAColumns * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(deviceB, hostB, numBRows * numBColumns * sizeof(float), hipMemcpyHostToDevice);

    //@@ Initialize the grid and block dimensions here DONE
    dim3 DimGrid(ceil(numCColumns / BLOCK_WIDTH), ceil(numCRows / BLOCK_WIDTH), 1);
    dim3 DimBlock(BLOCK_WIDTH, BLOCK_WIDTH, 1);

    //@@ Launch the GPU Kernel here DONE
    matrixMultiply<<<DimGrid, DimBlock>>>(deviceA, deviceB, deviceC,
                                          numARows, numAColumns,
                                          numBRows, numBColumns,
                                          numCRows, numCColumns);

    hipDeviceSynchronize();

    //@@ Copy the GPU memory back to the CPU here DONE
    hipMemcpy(hostC, deviceC, numCRows * numCColumns * sizeof(float), hipMemcpyDeviceToHost);

    //@@ Free the GPU memory here DONE
    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);

    wbSolution(args, hostC, numCRows, numCColumns);

    free(hostA);
    free(hostB);
    free(hostC);

    return 0;
}
